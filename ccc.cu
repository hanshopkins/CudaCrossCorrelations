#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "/usr/include/hip/hip_runtime.h"
#include "/usr/include/"

__global__ void CCKernel(float* d_vecA_re, float* d_vecA_im, float* d_vecB_re, float* d_vecB_im, unsigned int len, float* d_multvalues_re, float* d_multvalues_im, int startShift, unsigned int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x; //this i value is the position in the A vector
	int j = blockDim.y * blockIdx.y + threadIdx.y; //the j value is the shift
	
	if ((i < len) && (j < N)) //making sure this thread is in the matrix
	{
		unsigned int pos = j * len + i; //going left to right then top to bottom

		int B_Index = i + startShift + j; //the B coordinate we're taking for this particular thread
		
		if ((B_Index >= 0) && (B_Index < len)) //this if checks whether the shifted position is still inside the B vector
		{
			if (threadIdx.z == 0) //this splits up whether we're calculating the real or complex part. 0 is for real, 1 is for complex.
			{
				d_multvalues_re[pos] = d_vecA_re[i] * d_vecB_re[B_Index] + d_vecA_im[i] * d_vecB_im[B_Index]; //performing the complex conjugate multiplication for the real part
			}
			else
			{
				d_multvalues_im[pos] = - d_vecA_re[i] * d_vecB_im[B_Index] + d_vecA_im[i] * d_vecB_re[B_Index]; //performing the complex conjugate multiplication for the complex part
			}
		}
		else
		{
			if (threadIdx.z == 0)
			{
				d_multvalues_re[pos] = 0; //in this version of cross correlation, if they're shifted too far, the value is set to 0 instead of wrapping around
			}
			else
			{
				d_multvalues_im[pos] = 0;
			}
		}
	}
}

__global__ void sumRows(float* d_multValues_re, float* d_multValues_im, float* d_outputVec, unsigned int len, unsigned int N)
{
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x; //i is the row, where even corresponds to a real row and odd corresponds to an imaginary row.

	if (i < 2 * N)
	{
		float sum = 0;
		if (i % 2 == 0) //splitting between the even real case and the odd imaginary case
		{
			for (unsigned int j = 0; j < len; j++) //the actual summing loop
			{
				sum = sum + d_multValues_re[i/2 * len + j];
			}
		}
		else
		{
			for (unsigned int j = 0; j < len; j++)
			{
				sum = sum + d_multValues_im[(i-1) / 2 * len + j];
			}
		}
		sum = sum/len; //normalization
		d_outputVec[i] = sum;
	}
}

extern "C" {
	void cross_correlations(const float* vecA_re, const float* vecA_im, const float* vecB_re, const float* vecB_im, const unsigned int len, const int startShift, const int stopShift, float* outputVec) //len is the length of the input vectors
	{
		if (stopShift >= startShift)
		{
			unsigned int N = stopShift - startShift + 1; //N is how many shifting values you're checking. The output vector will contain 2N floats since it's complex.

			//allocating memory on the decide for both input vectors and copying them over
			size_t inputSize = len * sizeof(float);
			float* d_vecA_re = NULL;
			hipMalloc((void**)&d_vecA_re, inputSize);
			hipMemcpy(d_vecA_re, vecA_re, inputSize, hipMemcpyHostToDevice);

			float* d_vecA_im = NULL;
			hipMalloc((void**)&d_vecA_im, inputSize);
			hipMemcpy(d_vecA_im, vecA_im, inputSize, hipMemcpyHostToDevice);

			float* d_vecB_re = NULL;
			hipMalloc((void**)&d_vecB_re, inputSize);
			hipMemcpy(d_vecB_re, vecB_re, inputSize, hipMemcpyHostToDevice);

			float* d_vecB_im = NULL;
			hipMalloc((void**)&d_vecB_im, inputSize);
			hipMemcpy(d_vecB_im, vecB_im, inputSize, hipMemcpyHostToDevice);

			float* d_multValues_re = NULL;
			hipMalloc((void**)&d_multValues_re, len * N * sizeof(float));

			float* d_multValues_im = NULL;
			hipMalloc((void**)&d_multValues_im, len * N * sizeof(float));

			/*finding the kernel dimensions
			*
			* The multiplication values matrix is going to look like this
			* [[,,, ... ,,,] <- each row contains all the multiplications for each shift, so this would be shit by startShift
			* [,,, ... ,,,] <- shift by startShift + 1
			* ...
			* [,,, ... ,,,] <- shift by endShift - 1
			* [,,, ... ,,,]] <- shift by endShift
			*
			* so there are N rows and len colums. z = 0 is the real value, and z = 1 is the imaginary value.
			*
			*/

			dim3 dimBlock(16, 16, 2);
			dim3 dimGrid((len + 15) / 16, (N + 15) / 16, 1);

			hipError_t err = hipSuccess;
			CCKernel <<<dimGrid, dimBlock>>> (d_vecA_re, d_vecA_im, d_vecB_re, d_vecB_im, len, d_multValues_re, d_multValues_im, startShift, N);
			err = hipGetLastError();

			if (err != hipSuccess)
			{
				fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}

			
			/*float* h_mvaluestodisplay = (float*) malloc(len * N * sizeof(float));
			hipMemcpy(h_mvaluestodisplay, d_multValues_re, len * N * sizeof(float), hipMemcpyDeviceToHost);
			for (int i = 0; i < len * N; i++)
			{
				printf("%f ", h_mvaluestodisplay[i]);
				if ((i+1)%len == 0)
				{
					printf("\n");
				}
			}
			printf("\n");*/

			//we don't need the input vectors on the device anymore
			hipFree(d_vecA_re);
			hipFree(d_vecA_im);
			hipFree(d_vecB_re);
			hipFree(d_vecB_im);

			//creating the output vector. It contains 2N floats since each value is complex.
			size_t outputSize = 2 * N * sizeof(float);
			float* d_outputVec = NULL;
			hipMalloc((void**)&d_outputVec, outputSize);

			//finally summing all the values into the output vec. I'm going to sort this like [0_real, 0_imaginary, 1_real, 1_imaginary, ... (N-1)_real, (N-1)_imaginary]
			unsigned int threadsPerBlock = 256;
			unsigned int blocksPerGrid = (2 * N + threadsPerBlock - 1) / threadsPerBlock;
			sumRows <<<threadsPerBlock, blocksPerGrid>>> (d_multValues_re, d_multValues_im, d_outputVec, len, N);

			//freeing up these
			hipFree(d_multValues_re);
			hipFree(d_multValues_im);

			//copying the output vector back to the host
			hipMemcpy(outputVec, d_outputVec, outputSize, hipMemcpyDeviceToHost);
			hipFree(d_outputVec);
		}
		else
		{
			printf("Error: Starting shift is higher than the stopping shift.");
		}
	}
}
